/* -*- mode: c++ -*- */
#include <hip/hip_runtime.h>
#include "gg.h"

static struct ggc_rt_dev_info dinfo = {-1, -1};

void ggc_init_dev_info() {
  int dev;
  struct hipDeviceProp_t p;

  check_cuda(hipGetDevice(&dev));
  dinfo.dev = dev;
  
  check_cuda(hipGetDeviceProperties(&p, dev));
  dinfo.nSM = p.multiProcessorCount;
}

void ggc_set_gpu_device(int dev) {
  check_cuda(hipSetDevice(dev));
  ggc_init_dev_info();
}

int ggc_get_nSM() {
  if(dinfo.dev == -1)
    ggc_init_dev_info();

  return dinfo.nSM;
}
