#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 * This file belongs to the Galois project, a C++ library for exploiting parallelism.
 * The code is being released under the terms of the 3-Clause BSD License (a
 * copy is located in LICENSE.txt at the top-level directory).
 *
 * Copyright (C) 2018, The University of Texas at Austin. All rights reserved.
 * UNIVERSITY EXPRESSLY DISCLAIMS ANY AND ALL WARRANTIES CONCERNING THIS
 * SOFTWARE AND DOCUMENTATION, INCLUDING ANY WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR ANY PARTICULAR PURPOSE, NON-INFRINGEMENT AND WARRANTIES OF
 * PERFORMANCE, AND ANY WARRANTY THAT MIGHT OTHERWISE ARISE FROM COURSE OF
 * DEALING OR USAGE OF TRADE.  NO WARRANTY IS EITHER EXPRESS OR IMPLIED WITH
 * RESPECT TO THE USE OF THE SOFTWARE OR DOCUMENTATION. Under no circumstances
 * shall University be liable for incidental, special, indirect, direct or
 * consequential damages or loss of profits, interruption of business, or
 * related expenses which may arise from use of Software or Documentation,
 * including but not limited to those resulting from defects in Software and/or
 * Documentation, or loss or inaccuracy of data of any kind.
 */

/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
#include "kernels/reduce.cuh"
#include "kcore_pull_cuda.cuh"
static const int __tb_KCore = TB_SIZE;
__global__ void DegreeCounting(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, DynamicBitset * bitset_current_degree)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_current_degree[src] = graph.getOutDegree(src);
      bitset_current_degree->set(src);
    }
  }
  // FP: "8 -> 9;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, uint8_t * p_flag, uint8_t * p_pull_flag, uint32_t * p_trim)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_flag[src] = true;
      p_trim[src] = 0;
      p_current_degree[src] = 0;
      p_pull_flag[src] = false;
    }
  }
  // FP: "10 -> 11;
}
__global__ void LiveUpdate(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_k_core_num, uint32_t * p_current_degree, uint8_t * p_flag, uint8_t * p_pull_flag, uint32_t * p_trim, HGAccumulator<unsigned int> DGAccumulator_accum)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage DGAccumulator_accum_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_accum.thread_entry();
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      if (p_flag[src])
      {
        if (p_trim[src] > 0)
        {
          p_current_degree[src] = p_current_degree[src] - p_trim[src];
        }
        if (p_current_degree[src] < local_k_core_num)
        {
          p_flag[src] = false;
          DGAccumulator_accum.reduce( 1);
          p_pull_flag[src] = true;
        }
      }
      else
      {
        if (p_pull_flag[src])
        {
          p_pull_flag[src] = false;
        }
      }
      p_trim[src] = 0;
    }
  }
  // FP: "22 -> 23;
  DGAccumulator_accum.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(DGAccumulator_accum_ts);
  // FP: "23 -> 24;
}
__global__ void KCore(CSRGraph graph, unsigned int __begin, unsigned int __end, uint8_t * p_flag, uint8_t * p_pull_flag, uint32_t * p_trim, DynamicBitset * bitset_trim)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_KCore;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
      if (p_flag[src])
      {
      }
      else
      {
        pop = false;
      }
    }
    // FP: "12 -> 13;
    // FP: "15 -> 16;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "16 -> 17;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "17 -> 18;
    _np_closure[threadIdx.x].src = src;
    // FP: "18 -> 19;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "21 -> 22;
    // FP: "22 -> 23;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "23 -> 24;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "24 -> 25;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "27 -> 28;
    __syncthreads();
    // FP: "28 -> 29;
    while (true)
    {
      // FP: "29 -> 30;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "32 -> 33;
      __syncthreads();
      // FP: "33 -> 34;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "34 -> 35;
        __syncthreads();
        // FP: "35 -> 36;
        break;
      }
      // FP: "37 -> 38;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "40 -> 41;
      __syncthreads();
      // FP: "41 -> 42;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "42 -> 43;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "45 -> 46;
      //assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "46 -> 47;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type current_edge;
        current_edge = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(current_edge);
          if (p_pull_flag[dst])
          {
            atomicTestAdd(&p_trim[src], (uint32_t)1);
            bitset_trim->set(src);
          }
        }
      }
      // FP: "56 -> 57;
      __syncthreads();
    }
    // FP: "58 -> 59;

    // FP: "59 -> 60;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "60 -> 61;
      const int _np_laneid = hipcub::LaneId();
      // FP: "61 -> 62;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        //assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type current_edge;
          current_edge = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(current_edge);
            if (p_pull_flag[dst])
            {
              atomicTestAdd(&p_trim[src], (uint32_t)1);
              bitset_trim->set(src);
            }
          }
        }
      }
      // FP: "81 -> 82;
      __syncthreads();
      // FP: "82 -> 83;
    }

    // FP: "83 -> 84;
    __syncthreads();
    // FP: "84 -> 85;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "85 -> 86;
    while (_np.work())
    {
      // FP: "86 -> 87;
      int _np_i =0;
      // FP: "87 -> 88;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "88 -> 89;
      __syncthreads();
      // FP: "89 -> 90;

      // FP: "90 -> 91;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type current_edge;
        //assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        current_edge= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(current_edge);
          if (p_pull_flag[dst])
          {
            atomicTestAdd(&p_trim[src], (uint32_t)1);
            bitset_trim->set(src);
          }
        }
      }
      // FP: "101 -> 102;
      _np.execute_round_done(ITSIZE);
      // FP: "102 -> 103;
      __syncthreads();
    }
    // FP: "104 -> 105;
    //assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "106 -> 107;
}
__global__ void KCoreSanityCheck(CSRGraph graph, unsigned int __begin, unsigned int __end, uint8_t * p_flag, HGAccumulator<uint64_t> DGAccumulator_accum)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage DGAccumulator_accum_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_accum.thread_entry();
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      if (p_flag[src])
      {
        DGAccumulator_accum.reduce( 1);
      }
    }
  }
  // FP: "11 -> 12;
  DGAccumulator_accum.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(DGAccumulator_accum_ts);
  // FP: "12 -> 13;
}
void DegreeCounting_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  hipLaunchKernelGGL((DegreeCounting), dim3(blocks), dim3(threads), 0, 0, ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), (ctx->current_degree.is_updated.gpu_rd_ptr()));
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void DegreeCounting_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  DegreeCounting_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void DegreeCounting_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  DegreeCounting_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void DegreeCounting_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  DegreeCounting_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  hipLaunchKernelGGL((InitializeGraph), dim3(blocks), dim3(threads), 0, 0, ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->pull_flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void LiveUpdate_cuda(unsigned int  __begin, unsigned int  __end, unsigned int & DGAccumulator_accum, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<unsigned int> _DGAccumulator_accum;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<unsigned int> DGAccumulator_accumval  = Shared<unsigned int>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_accumval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_accum.rv = DGAccumulator_accumval.gpu_wr_ptr();
  // FP: "8 -> 9;
  hipLaunchKernelGGL((LiveUpdate), dim3(blocks), dim3(threads), 0, 0, ctx->gg, __begin, __end, local_k_core_num, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->pull_flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr(), _DGAccumulator_accum);
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  DGAccumulator_accum = *(DGAccumulator_accumval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void LiveUpdate_allNodes_cuda(unsigned int & DGAccumulator_accum, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  LiveUpdate_cuda(0, ctx->gg.nnodes, DGAccumulator_accum, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void LiveUpdate_masterNodes_cuda(unsigned int & DGAccumulator_accum, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  LiveUpdate_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_accum, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void LiveUpdate_nodesWithEdges_cuda(unsigned int & DGAccumulator_accum, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  LiveUpdate_cuda(0, ctx->numNodesWithEdges, DGAccumulator_accum, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void KCore_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  hipLaunchKernelGGL((KCore), dim3(blocks), dim3(__tb_KCore), 0, 0, ctx->gg, __begin, __end, ctx->flag.data.gpu_wr_ptr(), ctx->pull_flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr(), (ctx->trim.is_updated.gpu_rd_ptr()));
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void KCore_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCore_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void KCore_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCore_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void KCore_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCore_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_cuda(unsigned int  __begin, unsigned int  __end, uint64_t & DGAccumulator_accum, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<uint64_t> _DGAccumulator_accum;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<uint64_t> DGAccumulator_accumval  = Shared<uint64_t>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_accumval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_accum.rv = DGAccumulator_accumval.gpu_wr_ptr();
  // FP: "8 -> 9;
  hipLaunchKernelGGL((KCoreSanityCheck), dim3(blocks), dim3(threads), 0, 0, ctx->gg, __begin, __end, ctx->flag.data.gpu_wr_ptr(), _DGAccumulator_accum);
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  DGAccumulator_accum = *(DGAccumulator_accumval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void KCoreSanityCheck_allNodes_cuda(uint64_t & DGAccumulator_accum, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(0, ctx->gg.nnodes, DGAccumulator_accum, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_masterNodes_cuda(uint64_t & DGAccumulator_accum, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_accum, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_nodesWithEdges_cuda(uint64_t & DGAccumulator_accum, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(0, ctx->numNodesWithEdges, DGAccumulator_accum, ctx);
  // FP: "2 -> 3;
}
